#include "hip/hip_runtime.h"
/*
 * This file is part of the VanitySearch distribution (https://github.com/JeanLucPons/VanitySearch).
 * Copyright (c) 2019 Jean Luc PONS.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef WIN64
#include <unistd.h>
#include <stdio.h>
#endif

#include "GPUEngine.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdint.h>
#include "../hash/sha256.h"
#include "../hash/ripemd160.h"
#include "../Timer.h"

#include "GPUGroup.h"
#include "GPUMath.h"
#include "GPUHash.h"
#include "GPUBase58.h"
#include "GPUWildcard.h"
#include "GPUCompute.h"

// ---------------------------------------------------------------------------------------

/*__global__ void comp_keys(uint32_t mode, address_t* address, uint32_t* lookup32, uint64_t* keys, uint32_t maxFound, uint32_t* found) {

	int xPtr = (blockIdx.x * blockDim.x) * 8;
	int yPtr = xPtr + 4 * blockDim.x;
	ComputeKeys(mode, keys + xPtr, keys + yPtr, address, lookup32, maxFound, found);

}

__global__ void comp_keys_p2sh(uint32_t mode, address_t* address, uint32_t* lookup32, uint64_t* keys, uint32_t maxFound, uint32_t* found) {

	int xPtr = (blockIdx.x * blockDim.x) * 8;
	int yPtr = xPtr + 4 * blockDim.x;
	ComputeKeysP2SH(mode, keys + xPtr, keys + yPtr, address, lookup32, maxFound, found);

}*/

// Andrew kernel, STEP_SIZE not used
__global__ void comp_keys_comp(address_t* sAddress, uint32_t* lookup32, uint64_t* keys, uint32_t* out) {

	int xPtr = (blockIdx.x * blockDim.x) * 8;
	int yPtr = xPtr + 4 * blockDim.x;

	uint64_t* startx = keys + xPtr;
	uint64_t* starty = keys + yPtr;

	uint64_t dx[GRP_SIZE / 2 + 1][4];
	uint64_t px[4];
	uint64_t py[4];
	uint64_t pyn[4];
	uint64_t sx[4];
	uint64_t sy[4];
	uint64_t dy[4];
	uint64_t _s[4];
	uint64_t _p2[4];

	uint32_t h[5];

	// Load starting key
	__syncthreads();
	Load256A(sx, startx);
	Load256A(sy, starty);		
	Load256(px, sx);
	Load256(py, sy);

	//for (uint32_t j = 0; j < STEP_SIZE / GRP_SIZE; j++) {

	// Fill group with delta x
	uint32_t i;
	for (i = 0; i < HSIZE; i++)
		ModSub256(dx[i], Gx[i], sx);
	ModSub256(dx[i], Gx[i], sx);  // For the first point
	ModSub256(dx[i + 1], _2Gnx, sx);  // For the next center point

	// Compute modular inverse
	_ModInvGrouped(dx);

	// We use the fact that P + i*G and P - i*G has the same deltax, so the same inverse
	// We compute key in the positive and negative way from the center of the group

	// Check starting point
	//CheckHashComp(sAddress, px, (uint8_t)(py[0] & 1), j * GRP_SIZE + (GRP_SIZE / 2), lookup32, maxFound, out);
	_GetHash160Comp(px, (uint8_t)(py[0] & 1), (uint8_t*)h);
	//CheckPoint(h, j * GRP_SIZE + (GRP_SIZE / 2), 0, true, sAddress, lookup32, maxFound, out, P2PKH);
	CheckPointCompLookupOnly(h, GRP_SIZE / 2, sAddress, lookup32, out);

	ModNeg256(pyn, py);

	for (i = 0; i < HSIZE; i++) {

		// P = StartPoint + i*G
		Load256(px, sx);
		Load256(py, sy);
		ModSub256(dy, Gy[i], py);

		_ModMult(_s, dy, dx[i]);      //  s = (p2.y-p1.y)*inverse(p2.x-p1.x)
		_ModSqr(_p2, _s);             // _p2 = pow2(s)

		ModSub256(px, _p2, px);
		ModSub256(px, Gx[i]);         // px = pow2(s) - p1.x - p2.x;

		ModSub256(py, Gx[i], px);
		_ModMult(py, _s);             // py = - s*(ret.x-p2.x)
		ModSub256(py, Gy[i]);         // py = - p2.y - s*(ret.x-p2.x);  

		//CheckHashComp(sAddress, px, (uint8_t)(py[0] & 1), j * GRP_SIZE + (GRP_SIZE / 2 + (i + 1)), lookup32, maxFound, out);
		_GetHash160Comp(px, (uint8_t)(py[0] & 1), (uint8_t*)h);
		//CheckPoint(h, j * GRP_SIZE + (GRP_SIZE / 2 + (i + 1)), 0, true, sAddress, lookup32, maxFound, out, P2PKH);
		CheckPointCompLookupOnly(h, GRP_SIZE / 2 + (i + 1), sAddress, lookup32, out);

		// P = StartPoint - i*G, if (x,y) = i*G then (x,-y) = -i*G
		Load256(px, sx);
		ModSub256(dy, pyn, Gy[i]);

		_ModMult(_s, dy, dx[i]);      //  s = (p2.y-p1.y)*inverse(p2.x-p1.x)
		_ModSqr(_p2, _s);             // _p = pow2(s)

		ModSub256(px, _p2, px);
		ModSub256(px, Gx[i]);         // px = pow2(s) - p1.x - p2.x;		

		ModSub256(py, px, Gx[i]);
		_ModMult(py, _s);             // py = s*(ret.x-p2.x)
		ModSub256(py, Gy[i], py);     // py = - p2.y - s*(ret.x-p2.x);

		//CheckHashComp(sAddress, px, (uint8_t)(py[0] & 1), j * GRP_SIZE + (GRP_SIZE / 2 - (i + 1)), lookup32, maxFound, out);
		_GetHash160Comp(px, (uint8_t)(py[0] & 1), (uint8_t*)h);
		//CheckPoint(h, j * GRP_SIZE + (GRP_SIZE / 2 - (i + 1)), 0, true, sAddress, lookup32, maxFound, out, P2PKH);
		CheckPointCompLookupOnly(h, GRP_SIZE / 2 - (i + 1), sAddress, lookup32, out);
	}

	// First point (startP - (GRP_SZIE/2)*G)
	Load256(px, sx);
	Load256(py, sy);
	ModNeg256(dy, Gy[i]);
	ModSub256(dy, py);

	_ModMult(_s, dy, dx[i]);      //  s = (p2.y-p1.y)*inverse(p2.x-p1.x)
	_ModSqr(_p2, _s);              // _p = pow2(s)

	ModSub256(px, _p2, px);
	ModSub256(px, Gx[i]);         // px = pow2(s) - p1.x - p2.x;	

	ModSub256(py, px, Gx[i]);
	_ModMult(py, _s);             // py = s*(ret.x-p2.x)
	ModSub256(py, Gy[i], py);     // py = - p2.y - s*(ret.x-p2.x);

	//CheckHashComp(sAddress, px, (uint8_t)(py[0] & 1), j * GRP_SIZE + (0), lookup32, maxFound, out);
	_GetHash160Comp(px, (uint8_t)(py[0] & 1), (uint8_t*)h);
	//CheckPoint(h, j * GRP_SIZE + (0), 0, true, sAddress, lookup32, maxFound, out, P2PKH);
	CheckPointCompLookupOnly(h, 0, sAddress, lookup32, out);

	i++;

	// Next start point (startP + GRP_SIZE*G)
	Load256(px, sx);
	Load256(py, sy);
	ModSub256(dy, _2Gny, py);

	_ModMult(_s, dy, dx[i]);      //  s = (p2.y-p1.y)*inverse(p2.x-p1.x)
	_ModSqr(_p2, _s);             // _p2 = pow2(s)

	ModSub256(px, _p2, px);
	ModSub256(px, _2Gnx);         // px = pow2(s) - p1.x - p2.x;

	ModSub256(py, _2Gnx, px);
	_ModMult(py, _s);             // py = - s*(ret.x-p2.x)
	ModSub256(py, _2Gny);         // py = - p2.y - s*(ret.x-p2.x);  

	//}

	// Update starting point
	__syncthreads();
	Store256A(startx, px);
	Store256A(starty, py);
}

/*__global__ void comp_keys_pattern(uint32_t mode, address_t* pattern, uint64_t* keys, uint32_t maxFound, uint32_t* found) {

	int xPtr = (blockIdx.x * blockDim.x) * 8;
	int yPtr = xPtr + 4 * blockDim.x;
	ComputeKeys(mode, keys + xPtr, keys + yPtr, NULL, (uint32_t*)pattern, maxFound, found);

}

__global__ void comp_keys_p2sh_pattern(uint32_t mode, address_t* pattern, uint64_t* keys, uint32_t maxFound, uint32_t* found) {

	int xPtr = (blockIdx.x * blockDim.x) * 8;
	int yPtr = xPtr + 4 * blockDim.x;
	ComputeKeysP2SH(mode, keys + xPtr, keys + yPtr, NULL, (uint32_t*)pattern, maxFound, found);

}*/

//#define FULLCHECK
#ifdef FULLCHECK

// ---------------------------------------------------------------------------------------

__global__ void chekc_mult(uint64_t* a, uint64_t* b, uint64_t* r) {

	_ModMult(r, a, b);
	r[4] = 0;

}

// ---------------------------------------------------------------------------------------

__global__ void chekc_hash160(uint64_t* x, uint64_t* y, uint32_t* h) {

	_GetHash160(x, y, (uint8_t*)h);
	_GetHash160Comp(x, y, (uint8_t*)(h + 5));

}

// ---------------------------------------------------------------------------------------

__global__ void get_endianness(uint32_t* endian) {

	uint32_t a = 0x01020304;
	uint8_t fb = *(uint8_t*)(&a);
	*endian = (fb == 0x04);

}

#endif //FULLCHECK

// ---------------------------------------------------------------------------------------

using namespace std;

std::string toHex(unsigned char* data, int length) {

	string ret;
	char tmp[3];
	for (int i = 0; i < length; i++) {
		if (i && i % 4 == 0) ret.append(" ");
		sprintf(tmp, "%02hhX", (int)data[i]);
		ret.append(tmp);
	}
	return ret;

}

int _ConvertSMVer2Cores(int major, int minor) {

	// Defines for GPU Architecture types (using the SM version to determine
	// the # of cores per SM
	typedef struct {
		int SM;  // 0xMm (hexidecimal notation), M = SM Major version,
		// and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] = {
		{0x60,  64},
		{0x61, 128},
		{0x62, 128},
		{0x70,  64},
		{0x72,  64},
		{0x75,  64},
		{0x80,  64},
		{0x86,  128},
		{0x89,  128},
		{0x90,  114},
		{-1, -1} };

	int index = 0;

	while (nGpuArchCoresPerSM[index].SM != -1) {
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
			return nGpuArchCoresPerSM[index].Cores;
		}

		index++;
	}

	return 0;
}

GPUEngine::GPUEngine(int nbThreadGroup, int nbThreadPerGroup, int gpuId, uint32_t maxFound) {

	// Initialise CUDA  
	initialised = false;
	hipError_t err;

	int numBlocks;

	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess) {
		fprintf(stderr, "GPUEngine: CudaGetDeviceCount %s\n", hipGetErrorString(error_id));
		return;
	}

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0) {
		fprintf(stderr, "GPUEngine: There are no available device(s) that support CUDA\n");
		return;
	}

	err = hipSetDevice(gpuId);
	if (err != hipSuccess) {
		fprintf(stderr, "GPUEngine: %s\n", hipGetErrorString(err));
		return;
	}

	// Andrew mod
	// set cpu spinwait flag to prevent 100% cpu usage
	err = hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
	if (err != hipSuccess) {
		fprintf(stderr, "GPUEngine: %s\n", hipGetErrorString(err));
		return;
	}

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, gpuId);

	//numBlocks = deviceProp.multiProcessorCount * 64;

	int numBlocksMin = deviceProp.multiProcessorCount * 64;
	numBlocks = 64;
	while (numBlocks <= numBlocksMin)
	{
		numBlocks *= 2;
	}

#ifdef _DEBUG
	numBlocks = 4;
#endif

	this->numThreadsGPU = numBlocks * NUM_THREADS_PER_BLOCK;
	this->maxFound = maxFound;
	this->outputSize = (maxFound * ITEM_SIZE + 4);

	char tmp[512];
	sprintf(tmp, "GPU #%d %s (%dx%d cores) Grid(%dx%d)",
		gpuId, deviceProp.name, deviceProp.multiProcessorCount,
		_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
		numThreadsGPU / NUM_THREADS_PER_BLOCK, NUM_THREADS_PER_BLOCK);
	deviceName = std::string(tmp);

	// Allocate memory
	err = hipMalloc((void**)&inputAddress, _64K * 2);
	if (err != hipSuccess) {
		fprintf(stderr, "GPUEngine: Allocate address memory: %s\n", hipGetErrorString(err));
		return;
	}
	err = hipHostAlloc(&inputAddressPinned, _64K * 2, hipHostMallocWriteCombined | hipHostMallocMapped);
	if (err != hipSuccess) {
		fprintf(stderr, "GPUEngine: Allocate address pinned memory: %s\n", hipGetErrorString(err));
		return;
	}
	err = hipMalloc((void**)&inputKey, numThreadsGPU * 32 * 2);
	if (err != hipSuccess) {
		fprintf(stderr, "GPUEngine: Allocate input memory: %s\n", hipGetErrorString(err));
		return;
	}
	err = hipHostAlloc(&inputKeyPinned, numThreadsGPU * 32 * 2, hipHostMallocWriteCombined | hipHostMallocMapped);
	if (err != hipSuccess) {
		fprintf(stderr, "GPUEngine: Allocate input pinned memory: %s\n", hipGetErrorString(err));
		return;
	}
	err = hipMalloc((void**)&outputBuffer, outputSize);
	if (err != hipSuccess) {
		fprintf(stderr, "GPUEngine: Allocate output memory: %s\n", hipGetErrorString(err));
		return;
	}
	err = hipHostAlloc(&outputBufferPinned, outputSize, hipHostMallocMapped);
	if (err != hipSuccess) {
		fprintf(stderr, "GPUEngine: Allocate output pinned memory: %s\n", hipGetErrorString(err));
		return;
	}

	searchMode = SEARCH_COMPRESSED;
	searchType = P2PKH;
	initialised = true;
	pattern = "";
	hasPattern = false;
	inputAddressLookUp = NULL;
}

int GPUEngine::GetGroupSize() {
	return GRP_SIZE;
}

void GPUEngine::PrintCudaInfo() {

	hipError_t err;

	const char* sComputeMode[] =
	{
	  "Multiple host threads",
	  "Only one host thread",
	  "No host thread",
	  "Multiple process threads",
	  "Unknown",
	   NULL
	};

	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess) {
		fprintf(stderr, "GPUEngine: CudaGetDeviceCount %s\n", hipGetErrorString(error_id));
		return;
	}

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0) {
		fprintf(stderr, "GPUEngine: There are no available device(s) that support CUDA\n");
		return;
	}

	for (int i = 0; i < deviceCount; i++) {

		err = hipSetDevice(i);
		if (err != hipSuccess) {
			fprintf(stderr, "GPUEngine: hipSetDevice(%d) %s\n", i, hipGetErrorString(err));
			return;
		}

		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, i);
		fprintf(stdout, "GPU #%d %s (%dx%d cores) (Cap %d.%d) (%.1f MB) (%s)\n",
			i, deviceProp.name, deviceProp.multiProcessorCount,
			_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
			deviceProp.major, deviceProp.minor, (double)deviceProp.totalGlobalMem / 1048576.0,
			sComputeMode[deviceProp.computeMode]);
	}
}

GPUEngine::~GPUEngine() {

	hipFree(inputKey);
	hipFree(inputAddress);
	if (inputAddressLookUp) hipFree(inputAddressLookUp);
	hipHostFree(outputBufferPinned);
	hipFree(outputBuffer);
}

int GPUEngine::GetNumThreadsGPU() {
	return numThreadsGPU;
}

void GPUEngine::SetSearchMode(int searchMode) {
	this->searchMode = searchMode;
}

void GPUEngine::SetSearchType(int searchType) {
	this->searchType = searchType;
}

void GPUEngine::SetAddress(std::vector<address_t> addresses) {

	memset(inputAddressPinned, 0, _64K * 2);
	for (int i = 0; i < (int)addresses.size(); i++)
		inputAddressPinned[addresses[i]] = 1;

	// Fill device memory
	hipMemcpy(inputAddress, inputAddressPinned, _64K * 2, hipMemcpyHostToDevice);

	// We do not need the input pinned memory anymore
	hipHostFree(inputAddressPinned);
	inputAddressPinned = NULL;
	lostWarning = false;

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "GPUEngine: SetAddress: %s\n", hipGetErrorString(err));
	}
}

void GPUEngine::SetPattern(const char* pattern) {

	strcpy((char*)inputAddressPinned, pattern);

	// Fill device memory
	hipMemcpy(inputAddress, inputAddressPinned, _64K * 2, hipMemcpyHostToDevice);

	// We do not need the input pinned memory anymore
	hipHostFree(inputAddressPinned);
	inputAddressPinned = NULL;
	lostWarning = false;

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "GPUEngine: SetPattern: %s\n", hipGetErrorString(err));
	}

	hasPattern = true;
}

void GPUEngine::SetAddress(std::vector<LADDRESS> addresses, uint32_t totalAddress) {

	// Allocate memory for the second level of lookup tables
	hipError_t err = hipMalloc((void**)&inputAddressLookUp, (_64K + totalAddress) * 4);
	if (err != hipSuccess) {
		fprintf(stderr, "GPUEngine: Allocate address lookup memory: %s\n", hipGetErrorString(err));
		return;
	}
	err = hipHostAlloc(&inputAddressLookUpPinned, (_64K + totalAddress) * 4, hipHostMallocWriteCombined | hipHostMallocMapped);
	if (err != hipSuccess) {
		fprintf(stderr, "GPUEngine: Allocate address lookup pinned memory: %s\n", hipGetErrorString(err));
		return;
	}

	uint32_t offset = _64K;
	memset(inputAddressPinned, 0, _64K * 2);
	memset(inputAddressLookUpPinned, 0, _64K * 4);
	for (int i = 0; i < (int)addresses.size(); i++) {
		int nbLAddress = (int)addresses[i].lAddresses.size();
		inputAddressPinned[addresses[i].sAddress] = (uint16_t)nbLAddress;
		inputAddressLookUpPinned[addresses[i].sAddress] = offset;
		for (int j = 0; j < nbLAddress; j++) {
			inputAddressLookUpPinned[offset++] = addresses[i].lAddresses[j];
		}
	}

	if (offset != (_64K + totalAddress)) {
		fprintf(stderr, "GPUEngine: Wrong totalAddress %d!=%d!\n", offset - _64K, totalAddress);
		return;
	}

	// Fill device memory
	hipMemcpy(inputAddress, inputAddressPinned, _64K * 2, hipMemcpyHostToDevice);
	hipMemcpy(inputAddressLookUp, inputAddressLookUpPinned, (_64K + totalAddress) * 4, hipMemcpyHostToDevice);

	// We do not need the input pinned memory anymore
	hipHostFree(inputAddressPinned);
	inputAddressPinned = NULL;
	hipHostFree(inputAddressLookUpPinned);
	inputAddressLookUpPinned = NULL;
	lostWarning = false;

	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "GPUEngine: SetAddress (large): %s\n", hipGetErrorString(err));
	}
}

bool GPUEngine::callKernel() {

	// Reset nbFound
	hipMemset(outputBuffer, 0, 4);
			
	comp_keys_comp << < numThreadsGPU / NUM_THREADS_PER_BLOCK, NUM_THREADS_PER_BLOCK >> >
		(inputAddress, inputAddressLookUp, inputKey, outputBuffer);		

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "GPUEngine: Kernel: %s\n", hipGetErrorString(err));
		return false;
	}
	return true;
}

bool GPUEngine::SetKeys(Point* p) {

	// Sets the starting keys for each thread
	// p must contains numThreadsGPU public keys
	for (int i = 0; i < numThreadsGPU; i += NUM_THREADS_PER_BLOCK) {
		for (int j = 0; j < NUM_THREADS_PER_BLOCK; j++) {

			inputKeyPinned[8 * i + j + 0 * NUM_THREADS_PER_BLOCK] = p[i + j].x.bits64[0];
			inputKeyPinned[8 * i + j + 1 * NUM_THREADS_PER_BLOCK] = p[i + j].x.bits64[1];
			inputKeyPinned[8 * i + j + 2 * NUM_THREADS_PER_BLOCK] = p[i + j].x.bits64[2];
			inputKeyPinned[8 * i + j + 3 * NUM_THREADS_PER_BLOCK] = p[i + j].x.bits64[3];

			inputKeyPinned[8 * i + j + 4 * NUM_THREADS_PER_BLOCK] = p[i + j].y.bits64[0];
			inputKeyPinned[8 * i + j + 5 * NUM_THREADS_PER_BLOCK] = p[i + j].y.bits64[1];
			inputKeyPinned[8 * i + j + 6 * NUM_THREADS_PER_BLOCK] = p[i + j].y.bits64[2];
			inputKeyPinned[8 * i + j + 7 * NUM_THREADS_PER_BLOCK] = p[i + j].y.bits64[3];
		}
	}

	// Fill device memory
	hipMemcpy(inputKey, inputKeyPinned, numThreadsGPU * 32 * 2, hipMemcpyHostToDevice);

	// We do not need the input pinned memory anymore
	hipHostFree(inputKeyPinned);
	inputKeyPinned = NULL;

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "GPUEngine: SetKeys: %s\n", hipGetErrorString(err));
	}

	return callKernel();
}

bool GPUEngine::Launch(std::vector<ITEM>& addressFound, bool spinWait) {

	addressFound.clear();

	// Get the result
	if (spinWait) {

		hipMemcpy(outputBufferPinned, outputBuffer, outputSize, hipMemcpyDeviceToHost);
	}
	else {

		// Use hipMemcpyAsync to avoid default spin wait of hipMemcpy wich takes 100% CPU
		hipEvent_t evt;
		hipEventCreate(&evt);

		//hipMemcpy(outputAddressPinned, outputAddress, 4, hipMemcpyDeviceToHost);
		hipMemcpyAsync(outputBufferPinned, outputBuffer, 4, hipMemcpyDeviceToHost, 0);

		hipEventRecord(evt, 0);
		while (hipEventQuery(evt) == hipErrorNotReady) {
			// Sleep 1 ms to free the CPU
			Timer::SleepMillis(1);
		}
		hipEventDestroy(evt);
	}

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "GPUEngine: Launch: %s\n", hipGetErrorString(err));
		return false;
	}

	// Look for address found
	uint32_t nbFound = outputBufferPinned[0];
	if (nbFound > maxFound) {
		// address has been lost
		if (!lostWarning) {
			fprintf(stdout, "\nWarning, %d items lost\nHint: Search with less addresses, less threads (-g) or increase maxFound (-m)\n", (nbFound - maxFound));
			lostWarning = true;
		}
		nbFound = maxFound;
	}

	// When can perform a standard copy, the kernel is eneded
	hipMemcpy(outputBufferPinned, outputBuffer, nbFound * ITEM_SIZE + 4, hipMemcpyDeviceToHost);

	for (uint32_t i = 0; i < nbFound; i++) {
		uint32_t* itemPtr = outputBufferPinned + (i * ITEM_SIZE32 + 1);
		ITEM it;
		it.thId = itemPtr[0];
		int16_t* ptr = (int16_t*)&(itemPtr[1]);
		it.endo = ptr[0] & 0x7FFF;
		it.mode = (ptr[0] & 0x8000) != 0;
		it.incr = ptr[1];
		it.hash = (uint8_t*)(itemPtr + 2);
		addressFound.push_back(it);
	}

	return callKernel();
}

bool GPUEngine::CheckHash(uint8_t* h, vector<ITEM>& found, int tid, int incr, int endo, int* nbOK) {

	bool ok = true;

	// Search in found by GPU
	bool f = false;
	int l = 0;
	//printf("Search: %s\n", toHex(h,20).c_str());
	while (l < found.size() && !f) {
		f = ripemd160_comp_hash(found[l].hash, h);
		if (!f) l++;
	}
	if (f) {
		found.erase(found.begin() + l);
		*nbOK = *nbOK + 1;
	}
	else {
		ok = false;
		fprintf(stdout, "Expected item not found %s (thread=%d, incr=%d, endo=%d)\n",
			toHex(h, 20).c_str(), tid, incr, endo);
		if (found[l].hash != NULL)
			fprintf(stdout, "%s\n", toHex(found[l].hash, 20).c_str());
		else
			fprintf(stdout, "NULL\n");
	}

	return ok;
}

bool GPUEngine::Check(Secp256K1* secp) {

	uint8_t h[20];
	int i = 0;
	int j = 0;
	bool ok = true;

	if (!initialised)
		return false;

	fprintf(stdout, "GPU: %s\n", deviceName.c_str());

#ifdef FULLCHECK

	// Get endianess
	get_endianness << <1, 1 >> > (outputAddress);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("GPUEngine: get_endianness: %s\n", hipGetErrorString(err));
		return false;
	}
	hipMemcpy(outputAddressPinned, outputAddress, 1, hipMemcpyDeviceToHost);
	littleEndian = *outputAddressPinned != 0;
	printf("Endianness: %s\n", (littleEndian ? "Little" : "Big"));

	// Check modular mult
	Int a;
	Int b;
	Int r;
	Int c;
	a.Rand(256);
	b.Rand(256);
	c.ModMulK1(&a, &b);
	memcpy(inputKeyPinned, a.bits64, BIFULLSIZE);
	memcpy(inputKeyPinned + 5, b.bits64, BIFULLSIZE);
	hipMemcpy(inputKey, inputKeyPinned, BIFULLSIZE * 2, hipMemcpyHostToDevice);
	chekc_mult << <1, 1 >> > (inputKey, inputKey + 5, (uint64_t*)outputAddress);
	hipMemcpy(outputAddressPinned, outputAddress, BIFULLSIZE, hipMemcpyDeviceToHost);
	memcpy(r.bits64, outputAddressPinned, BIFULLSIZE);

	if (!c.IsEqual(&r)) {
		printf("\nModular Mult wrong:\nR=%s\nC=%s\n",
			toHex((uint8_t*)r.bits64, BIFULLSIZE).c_str(),
			toHex((uint8_t*)c.bits64, BIFULLSIZE).c_str());
		return false;
	}

	// Check hash 160C
	uint8_t hc[20];
	Point pi;
	pi.x.Rand(256);
	pi.y.Rand(256);
	secp.GetHash160(pi, false, h);
	secp.GetHash160(pi, true, hc);
	memcpy(inputKeyPinned, pi.x.bits64, BIFULLSIZE);
	memcpy(inputKeyPinned + 5, pi.y.bits64, BIFULLSIZE);
	hipMemcpy(inputKey, inputKeyPinned, BIFULLSIZE * 2, hipMemcpyHostToDevice);
	chekc_hash160 << <1, 1 >> > (inputKey, inputKey + 5, outputAddress);
	hipMemcpy(outputAddressPinned, outputAddress, 64, hipMemcpyDeviceToHost);

	if (!ripemd160_comp_hash((uint8_t*)outputAddressPinned, h)) {
		printf("\nGetHask160 wrong:\n%s\n%s\n",
			toHex((uint8_t*)outputAddressPinned, 20).c_str(),
			toHex(h, 20).c_str());
		return false;
	}
	if (!ripemd160_comp_hash((uint8_t*)(outputAddressPinned + 5), hc)) {
		printf("\nGetHask160Comp wrong:\n%s\n%s\n",
			toHex((uint8_t*)(outputAddressPinned + 5), 20).c_str(),
			toHex(h, 20).c_str());
		return false;
	}

#endif //FULLCHECK

	Point* p = new Point[numThreadsGPU];
	Point* p2 = new Point[numThreadsGPU];
	Int k;

	// Check kernel
	int nbFoundCPU[6];
	int nbOK[6];
	vector<ITEM> found;
	bool searchComp;

	if (searchMode == SEARCH_BOTH) {
		fprintf(stdout, "Warning, Check function does not support BOTH_MODE, use either compressed or uncompressed");
		return true;
	}

	searchComp = (searchMode == SEARCH_COMPRESSED) ? true : false;

	uint32_t seed = (uint32_t)time(NULL);
	fprintf(stdout, "Seed: %u\n", seed);
	rseed(seed);
	memset(nbOK, 0, sizeof(nbOK));
	memset(nbFoundCPU, 0, sizeof(nbFoundCPU));
	for (int i = 0; i < numThreadsGPU; i++) {
		k.Rand(64);
		p[i] = secp->ComputePublicKey(&k);
		// Group starts at the middle
		k.Add((uint64_t)GRP_SIZE / 2);
		p2[i] = secp->ComputePublicKey(&k);
	}

	std::vector<address_t> prefs;
	prefs.push_back(0xFEFE);
	prefs.push_back(0x1234);
	SetAddress(prefs);
	SetKeys(p2);
	double t0 = Timer::get_tick();
	Launch(found, true);
	double t1 = Timer::get_tick();
	//Timer::printResult((char *)"Key", 6*STEP_SIZE*numThreadsGPU, t0, t1);
	Timer::printResult((char*)"Key", 1 * STEP_SIZE * numThreadsGPU, t0, t1);

	//for (int i = 0; i < found.size(); i++) {
	//  printf("[%d]: thId=%d incr=%d\n", i, found[i].thId,found[i].incr);
	//  printf("[%d]: %s\n", i,toHex(found[i].hash,20).c_str());
	//}

	fprintf(stdout, "ComputeKeys() found %d items , CPU check...\n", (int)found.size());

	//Int beta,beta2;
	//beta.SetBase16((char *)"7ae96a2b657c07106e64479eac3434e99cf0497512f58995c1396c28719501ee");
	//beta2.SetBase16((char *)"851695d49a83f8ef919bb86153cbcb16630fb68aed0a766a3ec693d68e6afa40");

	// Check with CPU
	for (j = 0; (j < numThreadsGPU); j++) {
		for (i = 0; i < STEP_SIZE; i++) {

			Point pt;//, pt1, pt2;
			pt = p[j];
			//pt1 = p[j];
			//pt2 = p[j];
			//pt1.x.ModMulK1(&beta);
			//pt2.x.ModMulK1(&beta2);
			p[j] = secp->NextKey(p[j]);

			// Point and endo
			secp->GetHash160(P2PKH, searchComp, pt, h);
			address_t pr = *(address_t*)h;
			if (pr == 0xFEFE || pr == 0x1234) {
				nbFoundCPU[0]++;
				ok &= CheckHash(h, found, j, i, 0, nbOK + 0);
			}
			/*
			secp->GetHash160(P2PKH, searchComp, pt1, h);
			pr = *(address_t *)h;
			if (pr == 0xFEFE || pr == 0x1234) {
			  nbFoundCPU[1]++;
			  ok &= CheckHash(h, found, j, i, 1, nbOK + 1);
			}
			secp->GetHash160(P2PKH, searchComp, pt2, h);
			pr = *(address_t *)h;
			if (pr == 0xFEFE || pr == 0x1234) {
			  nbFoundCPU[2]++;
			  ok &= CheckHash(h, found, j, i, 2, nbOK + 2);
			}

			// Symetrics
			pt.y.ModNeg();
			pt1.y.ModNeg();
			pt2.y.ModNeg();

			secp->GetHash160(P2PKH, searchComp, pt, h);
			pr = *(address_t *)h;
			if (pr == 0xFEFE || pr == 0x1234) {
			  nbFoundCPU[3]++;
			  ok &= CheckHash(h, found, j, -i, 0, nbOK + 3);
			}

			secp->GetHash160(P2PKH, searchComp, pt1, h);
			pr = *(address_t *)h;
			if (pr == 0xFEFE || pr == 0x1234) {
			  nbFoundCPU[4]++;
			  ok &= CheckHash(h, found, j, -i, 1, nbOK + 4);
			}
			secp->GetHash160(P2PKH, searchComp, pt2, h);
			pr = *(address_t *)h;
			if (pr == 0xFEFE || pr == 0x1234) {
			  nbFoundCPU[5]++;
			  ok &= CheckHash(h, found, j, -i, 2, nbOK + 5);
			}
			*/
		}
	}

	if (ok && found.size() != 0) {
		ok = false;
		fprintf(stdout, "Unexpected item found !\n");
	}

	if (!ok) {

		int nbF = nbFoundCPU[0] + nbFoundCPU[1] + nbFoundCPU[2] +
			nbFoundCPU[3] + nbFoundCPU[4] + nbFoundCPU[5];
		fprintf(stdout, "CPU found %d items\n", nbF);

		fprintf(stdout, "GPU: point   correct [%d/%d]\n", nbOK[0], nbFoundCPU[0]);
		/*
		printf("GPU: endo #1 correct [%d/%d]\n", nbOK[1] , nbFoundCPU[1]);
		printf("GPU: endo #2 correct [%d/%d]\n", nbOK[2] , nbFoundCPU[2]);

		printf("GPU: sym/point   correct [%d/%d]\n", nbOK[3] , nbFoundCPU[3]);
		printf("GPU: sym/endo #1 correct [%d/%d]\n", nbOK[4] , nbFoundCPU[4]);
		printf("GPU: sym/endo #2 correct [%d/%d]\n", nbOK[5] , nbFoundCPU[5]);
		*/
		fprintf(stdout, "GPU/CPU check Failed !\n");
	}

	if (ok) fprintf(stdout, "GPU/CPU check OK\n");

	delete[] p;
	delete[] p2;
	return ok;
}


